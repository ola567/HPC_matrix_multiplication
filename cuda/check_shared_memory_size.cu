#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        printf("No CUDA capable devices found\n");
        return 1;
    }

    // Select the first device
    int deviceId = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);

    printf("Shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);

    return 0;
}
