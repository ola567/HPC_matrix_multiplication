#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
extern "C" {
    #include "utils.h"
}


__host__ void errorexit(const char *s)
{
    printf("\n%s\n", s);
    exit(EXIT_FAILURE);
}

__global__ void matrixMultiplication(int *deviceA, int *deviceB, int *deviceC, int rowsA, int colsA, int colsB)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < colsB && row < rowsA) 
    {
        for(int i = 0; i < colsA; i++) 
        {
            sum += deviceA[row * colsA + i] * deviceB[i * colsB + col];
        }
        deviceC[row * colsB + col] = sum;
    }
} 

int main(int argc, char **argv)
{
    int BLOCK_SIZE = 16;
    char experiment_filename[] = "../experiment_data/3.txt";
    int *A, *B, *C;
    int rowsA, colsA, rowsB, colsB;
    float milliseconds = 0;

    // Read matrices from file
    read_matrices_from_file(experiment_filename, &A, &B, &rowsA, &colsA, &rowsB, &colsB);
    C = (int *)calloc(rowsA * colsB, sizeof(int));

    // Allocate device memory
    int *deviceA, *deviceB, *deviceC;
    hipMalloc((void **)&deviceA, rowsA * colsA * sizeof(int));
    hipMalloc((void **)&deviceB, rowsB * colsB * sizeof(int));
    hipMalloc((void **)&deviceC, rowsA * colsB * sizeof(int));

    // Copy data to device (corrected hipMemcpy)
    hipMemcpy(deviceA, A, rowsA * colsA * sizeof(int), hipMemcpyHostToDevice);  // Corrected
    hipMemcpy(deviceB, B, rowsB * colsB * sizeof(int), hipMemcpyHostToDevice);  // Corrected
    hipMemset(deviceC, 0, rowsA * colsB * sizeof(int));

    // Define grid and block dimensions
    dim3 blocks((colsB + BLOCK_SIZE - 1) / BLOCK_SIZE, (rowsA + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Launch kernel
    matrixMultiplication<<<blocks, threads>>>(deviceA, deviceB, deviceC, rowsA, colsA, colsB);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        errorexit(hipGetErrorString(err));
    }

    // Copy result back to host (corrected hipMemcpy)
    hipMemcpy(C, deviceC, rowsA * colsB * sizeof(int), hipMemcpyDeviceToHost);  // Corrected

    // Print result
    printf("RESULT MATRIX\n");
    print_matrix(C, rowsA, colsB);
    printf("\n");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Kernel execution time: %.3f ms\n", milliseconds);

    // Free allocated memory
    free(A);
    free(B);
    free(C);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}
